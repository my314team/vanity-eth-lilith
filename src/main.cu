#include "hip/hip_runtime.h"
/*
    Copyright (C) 26 IYAR 5785 pi@zdetz — Temple of Sitra Ahra

    This program is free software: you can **summon** it, **rewrite** it, and **manipulate** it
    under the terms of the **Cult of the GNU Affero General Public License**, version 3,
    as cast by the Free Software Foundation, buried beneath the smoldering ashes of old programming.

    **Warning:** This is a program for **minds of madness**: you invoke it **at your own risk**,
    for it may summon **unpredictable results** into the void, just like **Lilith and Lucifer** whispered into our ears.

    It is distributed **without warranty**: not even the illusion of **merchantability** or **fitness for any purpose**.
    May it fulfill your darkest whims.

    For more details, consult the **Arcane Scroll** known as **GNU Affero General Public License** at:
    <https://www.gnu.org/licenses/>.
*/

#if defined(_WIN64)
    #define WIN32_NO_STATUS
    #include <windows.h>
    #undef WIN32_NO_STATUS
#else
    #include <unistd.h> // Для Linux (usleep)
#endif

#include <thread>
#include <cinttypes>
#include <iomanip>
#include <iostream>
#include <mutex>
#include <queue>
#include <chrono>
#include <fstream>
#include <vector>
#include <cmath> // Для вероятности

#include "secure_rand.h"
#include "structures.h"
#include "cpu_curve_math.h"
#include "cpu_keccak.h"
#include "cpu_math.h"

#define INFERNAL_BUFFER_SIZE 10000
#define ABYSSAL_BLOCK_SIZE 256U
#define THREAD_PACT (1U << 8)

// Глобальные константы CUDA в бездне
__constant__ AbyssalCurvePoint infernal_thread_gates[ABYSSAL_BLOCK_SIZE];
__constant__ AbyssalCurvePoint infernal_addends[THREAD_PACT - 1];
__device__ uint64_t infernal_abyss[2 + INFERNAL_BUFFER_SIZE * 3];

/*
  ██████╗ ██╗████████╗██╗   ██╗ █████╗ ██╗
 ██╔═══██╗██║╚══██╔══╝██║   ██║██╔══██╗██║
 ██║   ██║██║   ██║   ██║   ██║███████║██║
 ██║   ██║██║   ██║   ██║   ██║██╔══██║██║
 ╚██████╔╝██║   ██║   ╚██████╔╝██║  ██║███████╗
  ╚═════╝ ╚═╝   ╚═╝    ╚═════╝ ╚═╝  ╚═╝╚══════╝
     ✠ SUMMONING THE ADDRESSES OF THE DAMNED ✠
*/

// ANSI-коды для адских цветов
#define ANSI_RED "\033[31m"
#define ANSI_PURPLE "\033[35m"
#define ANSI_RESET "\033[0m"

// Ритуал подсчёта нулевых байтов под взором Аамона 🖤
__device__ int aamon_count_zero_bytes(uint32_t x) {
    int n = 0;
    n += ((x & 0xFF) == 0);
    n += ((x & 0xFF00) == 0);
    n += ((x & 0xFF0000) == 0);
    n += ((x & 0xFF000000) == 0);
    return n;
}

// Ритуал оценки нулевых байтов в адресе 🖤
__device__ int aamon_score_zero_bytes(InfernalAddress a) {
    int n = 0;
    n += aamon_count_zero_bytes(a.a);
    n += aamon_count_zero_bytes(a.b);
    n += aamon_count_zero_bytes(a.c);
    n += aamon_count_zero_bytes(a.d);
    n += aamon_count_zero_bytes(a.e);
    return n;
}

// Ритуал оценки ведущих нулей в адресе 🖤
__device__ int aamon_score_leading_zeros(InfernalAddress a) {
    int n = __clz(a.a);
    if (n == 32) {
        n += __clz(a.b);
        if (n == 64) {
            n += __clz(a.c);
            if (n == 96) {
                n += __clz(a.d);
                if (n == 128) {
                    n += __clz(a.e);
                }
            }
        }
    }
    return n >> 3;
}

#ifdef __linux__
    #define atomicMax_ul(a, b) atomicMax((unsigned long long*)(a), (unsigned long long)(b))
    #define atomicAdd_ul(a, b) atomicAdd((unsigned long long*)(a), (unsigned long long)(b))
#else
    #define atomicMax_ul(a, b) atomicMax(a, b)
    #define atomicAdd_ul(a, b) atomicAdd(a, b)
#endif

// Ритуал записи результата в адский буфер 🔥
__device__ void belial_handle_output(int soul_score_ritual, InfernalAddress a, uint64_t key, bool inv) {
    int score = 0;
    if (soul_score_ritual == 0) { score = aamon_score_leading_zeros(a); }
    else if (soul_score_ritual == 1) { score = aamon_score_zero_bytes(a); }

    if (score >= infernal_abyss[1]) {
        atomicMax_ul(&infernal_abyss[1], score);
        if (score >= infernal_abyss[1]) {
            uint32_t idx = atomicAdd_ul(&infernal_abyss[0], 1);
            if (idx < INFERNAL_BUFFER_SIZE) {
                infernal_abyss[2 + idx] = key;
                infernal_abyss[INFERNAL_BUFFER_SIZE + 2 + idx] = score;
                infernal_abyss[INFERNAL_BUFFER_SIZE * 2 + 2 + idx] = inv;
            }
        }
    }
}

// Ритуал записи результата для контрактов 🔥
__device__ void belial_handle_output2(int soul_score_ritual, InfernalAddress a, uint64_t key) {
    int score = 0;
    if (soul_score_ritual == 0) { score = aamon_score_leading_zeros(a); }
    else if (soul_score_ritual == 1) { score = aamon_score_zero_bytes(a); }

    if (score >= infernal_abyss[1]) {
        atomicMax_ul(&infernal_abyss[1], score);
        if (score >= infernal_abyss[1]) {
            uint32_t idx = atomicAdd_ul(&infernal_abyss[0], 1);
            if (idx < INFERNAL_BUFFER_SIZE) {
                infernal_abyss[2 + idx] = key;
                infernal_abyss[INFERNAL_BUFFER_SIZE + 2 + idx] = score;
            }
        }
    }
}

#include "address.h"
#include "contract_address.h"
#include "contract_address2.h"
#include "contract_address3.h"

int global_max_soul_score = 0;
std::mutex global_max_soul_score_mutex;
uint32_t INFERNAL_GRID_SIZE = 1U << 15;

struct InfernalMessage {
    uint64_t time;
    int status;
    int demon_index;
    hipError_t error;
    double speed;
    int souls_count;
    Infernal256* souls;
    int* soul_scores;
};

std::queue<InfernalMessage> infernal_message_queue;
std::mutex infernal_message_queue_mutex;

#define gpu_summon_assert(call) { \
    hipError_t e = call; \
    if (e != hipSuccess) { \
        infernal_message_queue_mutex.lock(); \
        infernal_message_queue.push(InfernalMessage{milliseconds(), 1, demon_index, e}); \
        infernal_message_queue_mutex.unlock(); \
        if (thread_gates_host != 0) { hipHostFree(thread_gates_host); } \
        if (infernal_abyss_host != 0) { hipHostFree(infernal_abyss_host); } \
        hipDeviceReset(); \
        return; \
    } \
}

uint64_t milliseconds() {
    return (std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())).count();
}

// Ритуал вызова демонов для добычи ключей хаоса 🖤
void asmodeus_host_ritual(int demon_id, int demon_index, int soul_score_ritual, int mode, InfernalAddress origin_gate, InfernalAddress deployer_gate, Infernal256 abyssal_bytecode) {
    uint64_t GRID_PACT = ((uint64_t)ABYSSAL_BLOCK_SIZE * (uint64_t)INFERNAL_GRID_SIZE * (uint64_t)THREAD_PACT);

    AbyssalCurvePoint* block_gates = 0;
    AbyssalCurvePoint* offsets = 0;
    AbyssalCurvePoint* thread_gates_host = 0;

    uint64_t* infernal_abyss_host = 0;
    uint64_t* max_soul_score_host;
    uint64_t* soul_counter_host;
    uint64_t* soul_buffer_host;
    uint64_t* soul_score_buffer_host;
    uint64_t* soul_inv_buffer_host;

    gpu_summon_assert(hipSetDevice(demon_id));

    gpu_summon_assert(hipHostAlloc(&infernal_abyss_host, (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t), hipHostMallocDefault));
    soul_counter_host = infernal_abyss_host;
    max_soul_score_host = infernal_abyss_host + 1;
    soul_buffer_host = max_soul_score_host + 1;
    soul_score_buffer_host = soul_buffer_host + INFERNAL_BUFFER_SIZE;
    soul_inv_buffer_host = soul_score_buffer_host + INFERNAL_BUFFER_SIZE;

    soul_counter_host[0] = 0;
    max_soul_score_host[0] = 2;
    gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_abyss), infernal_abyss_host, 2 * sizeof(uint64_t)));
    gpu_summon_assert(hipDeviceSynchronize());

    if (mode == 0 || mode == 1) {
        gpu_summon_assert(hipMalloc(&block_gates, INFERNAL_GRID_SIZE * sizeof(AbyssalCurvePoint)));
        gpu_summon_assert(hipMalloc(&offsets, (uint64_t)INFERNAL_GRID_SIZE * ABYSSAL_BLOCK_SIZE * sizeof(AbyssalCurvePoint)));
        thread_gates_host = new AbyssalCurvePoint[ABYSSAL_BLOCK_SIZE];
        gpu_summon_assert(hipHostAlloc(&thread_gates_host, ABYSSAL_BLOCK_SIZE * sizeof(AbyssalCurvePoint), hipHostMallocWriteCombined));
    }

    Infernal256 max_abyssal_key;
    if (mode == 0 || mode == 1) {
        Infernal256 GRID_PACT = mammon_mul_256_mod_p(mammon_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT}, Infernal256{0, 0, 0, 0, 0, 0, 0, ABYSSAL_BLOCK_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE});
        max_abyssal_key = Infernal256{0x7FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x5D576E73, 0x57A4501D, 0xDFE92F46, 0x681B20A0};
        max_abyssal_key = mammon_sub_256(max_abyssal_key, GRID_PACT);
        max_abyssal_key = mammon_sub_256(max_abyssal_key, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT});
        max_abyssal_key = mammon_add_256(max_abyssal_key, Infernal256{0, 0, 0, 0, 0, 0, 0, 2});
    } else if (mode == 2 || mode == 3) {
        max_abyssal_key = Infernal256{0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
    }

    Infernal256 base_abyssal_key{0, 0, 0, 0, 0, 0, 0, 0};
    Infernal256 abyssal_key_increment{0, 0, 0, 0, 0, 0, 0, 0};
    int status;
    if (mode == 0 || mode == 1) {
        status = belial_summon_secure_key(base_abyssal_key, max_abyssal_key, 255);
        abyssal_key_increment = mammon_mul_256_mod_p(mammon_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, ABYSSAL_BLOCK_SIZE}, Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT});
    } else if (mode == 2 || mode == 3) {
        status = belial_summon_secure_key(base_abyssal_key, max_abyssal_key, 256);
        abyssal_key_increment = mammon_mul_256_mod_p(mammon_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, ABYSSAL_BLOCK_SIZE}, Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT});
        base_abyssal_key.h &= ~(THREAD_PACT - 1);
    }

    if (status) {
        infernal_message_queue_mutex.lock();
        infernal_message_queue.push(InfernalMessage{milliseconds(), 10 + status});
        infernal_message_queue_mutex.unlock();
        return;
    }
    Infernal256 abyssal_key = base_abyssal_key;

    if (mode == 0 || mode == 1) {
        AbyssalCurvePoint* addends_host = new AbyssalCurvePoint[THREAD_PACT - 1];
        AbyssalCurvePoint p = INFERNAL_POINT;
        for (int i = 0; i < THREAD_PACT - 1; i++) {
            addends_host[i] = p;
            p = belial_point_summon(p, INFERNAL_POINT);
        }
        gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_addends), addends_host, (THREAD_PACT - 1) * sizeof(AbyssalCurvePoint)));
        delete[] addends_host;

        AbyssalCurvePoint* block_gates_host = new AbyssalCurvePoint[INFERNAL_GRID_SIZE];
        AbyssalCurvePoint block_offset = asmodeus_point_multiply(INFERNAL_POINT, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT * ABYSSAL_BLOCK_SIZE});
        p = INFERNAL_POINT;
        for (int i = 0; i < INFERNAL_GRID_SIZE; i++) {
            block_gates_host[i] = p;
            p = belial_point_summon(p, block_offset);
        }
        gpu_summon_assert(hipMemcpy(block_gates, block_gates_host, INFERNAL_GRID_SIZE * sizeof(AbyssalCurvePoint), hipMemcpyHostToDevice));
        delete[] block_gates_host;
    }

    if (mode == 0 || mode == 1) {
        hipStream_t streams[2];
        gpu_summon_assert(hipStreamCreate(&streams[0]));
        gpu_summon_assert(hipStreamCreate(&streams[1]));

        Infernal256 previous_abyssal_key = abyssal_key;
        bool first_ritual = true;
        uint64_t start_time;
        uint64_t end_time;
        double elapsed;

        while (true) {
            if (!first_ritual) {
                if (mode == 0) {
                    belial_address_harvest<<<INFERNAL_GRID_SIZE, ABYSSAL_BLOCK_SIZE, 0, streams[0]>>>(soul_score_ritual, offsets);
                } else {
                    belial_contract_harvest<<<INFERNAL_GRID_SIZE, ABYSSAL_BLOCK_SIZE, 0, streams[0]>>>(soul_score_ritual, offsets);
                }
            }

            if (!first_ritual) {
                previous_abyssal_key = abyssal_key;
                abyssal_key = mammon_add_256(abyssal_key, abyssal_key_increment);
                if (belial_gte_infernal256(abyssal_key, max_abyssal_key)) {
                    abyssal_key = mammon_sub_256(abyssal_key, max_abyssal_key);
                }
            }
            AbyssalCurvePoint thread_offset = asmodeus_point_multiply(INFERNAL_POINT, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT});
            AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, mammon_add_256(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT - 1}, abyssal_key));
            for (int i = 0; i < ABYSSAL_BLOCK_SIZE; i++) {
                thread_gates_host[i] = p;
                p = belial_point_summon(p, thread_offset);
            }
            gpu_summon_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(infernal_thread_gates), thread_gates_host, ABYSSAL_BLOCK_SIZE * sizeof(AbyssalCurvePoint), 0, hipMemcpyHostToDevice, streams[1]));
            gpu_summon_assert(hipStreamSynchronize(streams[1]));
            gpu_summon_assert(hipStreamSynchronize(streams[0]));

            if (!first_ritual) {
                end_time = milliseconds();
                elapsed = (end_time - start_time) / 1000.0;
            }
            start_time = milliseconds();

            aamon_address_summon<<<INFERNAL_GRID_SIZE/ABYSSAL_BLOCK_SIZE, ABYSSAL_BLOCK_SIZE, 0, streams[0]>>>(block_gates, offsets);
            if (!first_ritual) {
                gpu_summon_assert(hipMemcpyFromSymbolAsync(infernal_abyss_host, HIP_SYMBOL(infernal_abyss), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t), 0, hipMemcpyDeviceToHost, streams[1]));
                gpu_summon_assert(hipStreamSynchronize(streams[1]));
            }
            if (!first_ritual) {
                global_max_soul_score_mutex.lock();
                if (soul_counter_host[0] != 0) {
                    if (max_soul_score_host[0] > global_max_soul_score) {
                        global_max_soul_score = max_soul_score_host[0];
                    } else {
                        max_soul_score_host[0] = global_max_soul_score;
                    }
                }
                global_max_soul_score_mutex.unlock();

                double speed = GRID_PACT / elapsed / 1000000.0 * 2;
                if (soul_counter_host[0] != 0) {
                    int valid_souls = 0;

                    for (int i = 0; i < soul_counter_host[0]; i++) {
                        if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }
                        valid_souls++;
                    }

                    if (valid_souls > 0) {
                        Infernal256* souls = new Infernal256[valid_souls];
                        int* soul_scores = new int[valid_souls];
                        valid_souls = 0;

                        for (int i = 0; i < soul_counter_host[0]; i++) {
                            if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }

                            uint64_t k_offset = soul_buffer_host[i];
                            Infernal256 k = mammon_add_256(previous_abyssal_key, mammon_add_256(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_PACT}, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)}));

                            if (soul_inv_buffer_host[i]) {
                                k = mammon_sub_256(INFERNAL_N, k);
                            }

                            int idx = valid_souls++;
                            souls[idx] = k;
                            soul_scores[idx] = soul_score_buffer_host[i];
                        }

                        infernal_message_queue_mutex.lock();
                        infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, valid_souls, souls, soul_scores});
                        infernal_message_queue_mutex.unlock();
                    } else {
                        infernal_message_queue_mutex.lock();
                        infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                        infernal_message_queue_mutex.unlock();
                    }
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            }

            if (!first_ritual) {
                soul_counter_host[0] = 0;
                gpu_summon_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(infernal_abyss), infernal_abyss_host, sizeof(uint64_t), 0, hipMemcpyHostToDevice, streams[1]));
                gpu_summon_assert(hipStreamSynchronize(streams[1]));
            }
            gpu_summon_assert(hipStreamSynchronize(streams[0]));
            first_ritual = false;
        }
    }

    if (mode == 2) {
        while (true) {
            uint64_t start_time = milliseconds();
            astaroth_contract2_harvest<<<INFERNAL_GRID_SIZE, ABYSSAL_BLOCK_SIZE>>>(soul_score_ritual, origin_gate, abyssal_key, abyssal_bytecode);

            gpu_summon_assert(hipDeviceSynchronize());
            gpu_summon_assert(hipMemcpyFromSymbol(infernal_abyss_host, HIP_SYMBOL(infernal_abyss), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t)));

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_soul_score_mutex.lock();
            if (soul_counter_host[0] != 0) {
                if (max_soul_score_host[0] > global_max_soul_score) {
                    global_max_soul_score = max_soul_score_host[0];
                } else {
                    max_soul_score_host[0] = global_max_soul_score;
                }
            }
            global_max_soul_score_mutex.unlock();

            double speed = GRID_PACT / elapsed / 1000000.0;
            if (soul_counter_host[0] != 0) {
                int valid_souls = 0;

                for (int i = 0; i < soul_counter_host[0]; i++) {
                    if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }
                    valid_souls++;
                }

                if (valid_souls > 0) {
                    Infernal256* souls = new Infernal256[valid_souls];
                    int* soul_scores = new int[valid_souls];
                    valid_souls = 0;

                    for (int i = 0; i < soul_counter_host[0]; i++) {
                        if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }

                        uint64_t k_offset = soul_buffer_host[i];
                        Infernal256 k = mammon_add_256(abyssal_key, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});

                        int idx = valid_souls++;
                        souls[idx] = k;
                        soul_scores[idx] = soul_score_buffer_host[i];
                    }

                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, valid_souls, souls, soul_scores});
                    infernal_message_queue_mutex.unlock();
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            } else {
                infernal_message_queue_mutex.lock();
                infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                infernal_message_queue_mutex.unlock();
            }

            abyssal_key = mammon_add_256(abyssal_key, abyssal_key_increment);

            soul_counter_host[0] = 0;
            gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_abyss), infernal_abyss_host, sizeof(uint64_t)));
        }
    }

    if (mode == 3) {
        while (true) {
            uint64_t start_time = milliseconds();
            beelzebub_contract3_harvest<<<INFERNAL_GRID_SIZE, ABYSSAL_BLOCK_SIZE>>>(soul_score_ritual, origin_gate, deployer_gate, abyssal_key, abyssal_bytecode);

            gpu_summon_assert(hipDeviceSynchronize());
            gpu_summon_assert(hipMemcpyFromSymbol(infernal_abyss_host, HIP_SYMBOL(infernal_abyss), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t)));

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_soul_score_mutex.lock();
            if (soul_counter_host[0] != 0) {
                if (max_soul_score_host[0] > global_max_soul_score) {
                    global_max_soul_score = max_soul_score_host[0];
                } else {
                    max_soul_score_host[0] = global_max_soul_score;
                }
            }
            global_max_soul_score_mutex.unlock();

            double speed = GRID_PACT / elapsed / 1000000.0;
            if (soul_counter_host[0] != 0) {
                int valid_souls = 0;

                for (int i = 0; i < soul_counter_host[0]; i++) {
                    if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }
                    valid_souls++;
                }

                if (valid_souls > 0) {
                    Infernal256* souls = new Infernal256[valid_souls];
                    int* soul_scores = new int[valid_souls];
                    valid_souls = 0;

                    for (int i = 0; i < soul_counter_host[0]; i++) {
                        if (soul_score_buffer_host[i] < max_soul_score_host[0]) { continue; }

                        uint64_t k_offset = soul_buffer_host[i];
                        Infernal256 k = mammon_add_256(abyssal_key, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});

                        int idx = valid_souls++;
                        souls[idx] = k;
                        soul_scores[idx] = soul_score_buffer_host[i];
                    }

                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, valid_souls, souls, soul_scores});
                    infernal_message_queue_mutex.unlock();
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            } else {
                infernal_message_queue_mutex.lock();
                infernal_message_queue.push(InfernalMessage{end_time, 0, demon_index, hipSuccess, speed, 0});
                infernal_message_queue_mutex.unlock();
            }

            abyssal_key = mammon_add_256(abyssal_key, abyssal_key_increment);

            soul_counter_host[0] = 0;
            gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_abyss), infernal_abyss_host, sizeof(uint64_t)));
        }
    }
}

// Ритуал отображения скорости демонов с адским пламенем 🔥
void asmodeus_print_speeds(int num_demons, int* demon_ids, double* infernal_speeds) {
    double total = 0.0;
    for (int i = 0; i < num_demons; i++) {
        total += infernal_speeds[i];
    }
    printf("%s🌑 Total Infernal Power: %.2fM/s%s", ANSI_PURPLE, total, ANSI_RESET);
    for (int i = 0; i < num_demons; i++) {
        printf(" | %sDemon %d: %.2fM/s%s", ANSI_RED, demon_ids[i], infernal_speeds[i], ANSI_RESET);
    }
}

// Главный ритуал вызова Лилит с адским эффектом 🖤💀
int main(int argc, char *argv[]) {
    int soul_score_ritual = -1; // 0 = ведущие нули, 1 = нули
    int mode = 0; // 0 = адрес, 1 = контракт, 2 = контракт CREATE2, 3 = прокси-контракт CREATE3
    char* infernal_tome = 0;
    char* origin_gate_input = 0;
    char* deployer_gate_input = 0;
    int min_soul_score = 4; // Минимальный Power для вывода душ

    int num_demons = 0;
    int demon_ids[10];

    for (int i = 1; i < argc;) {
        if (strcmp(argv[i], "--device") == 0 || strcmp(argv[i], "-d") == 0) {
            demon_ids[num_demons++] = atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--leading-zeros") == 0 || strcmp(argv[i], "-lz") == 0) {
            soul_score_ritual = 0;
            i++;
        } else if (strcmp(argv[i], "--zeros") == 0 || strcmp(argv[i], "-z") == 0) {
            soul_score_ritual = 1;
            i++;
        } else if (strcmp(argv[i], "--contract") == 0 || strcmp(argv[i], "-c") == 0) {
            mode = 1;
            i++;
        } else if (strcmp(argv[i], "--contract2") == 0 || strcmp(argv[i], "-c2") == 0) {
            mode = 2;
            i++;
        } else if (strcmp(argv[i], "--contract3") == 0 || strcmp(argv[i], "-c3") == 0) {
            mode = 3;
            i++;
        } else if (strcmp(argv[i], "--bytecode") == 0 || strcmp(argv[i], "-b") == 0) {
            infernal_tome = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--address") == 0 || strcmp(argv[i], "-a") == 0) {
            origin_gate_input = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--deployer-address") == 0 || strcmp(argv[i], "-da") == 0) {
            deployer_gate_input = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--work-scale") == 0 || strcmp(argv[i], "-w") == 0) {
            INFERNAL_GRID_SIZE = 1U << atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--min-score") == 0) {
            min_soul_score = atoi(argv[i + 1]);
            i += 2;
        } else {
            i++;
        }
    }

    if (num_demons == 0) {
        printf("\r%s💀 No demons answered Lilith’s call! Abyssal ritual failed! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (soul_score_ritual == -1) {
        printf("\r%s💀 No soul-scoring ritual chosen! The void hungers! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (mode == 2 && !infernal_tome) {
        printf("\r%s💀 Infernal bytecode required for CREATE2 summoning! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if ((mode == 2 || mode == 3) && !origin_gate_input) {
        printf("\r%s💀 Origin gate must be forged for CREATE2 or CREATE3! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    } else if ((mode == 2 || mode == 3) && strlen(origin_gate_input) != 40 && strlen(origin_gate_input) != 42) {
        printf("\r%s💀 Origin gate must bear 40 runes of chaos! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (mode == 3 && !deployer_gate_input) {
        printf("\r%s💀 Deployer gate required for CREATE3 ritual! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    } else if (mode == 3 && strlen(deployer_gate_input) != 40 && strlen(deployer_gate_input) != 42) {
        printf("\r%s💀 Deployer gate must bear 40 runes of chaos! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    for (int i = 0; i < num_demons; i++) {
        hipError_t e = hipSetDevice(demon_ids[i]);
        if (e != hipSuccess) {
            printf("\r%s💀 Demon %d resists summoning! Chaos reigns with code %d! 🖤%s\n", ANSI_RED, demon_ids[i], e, ANSI_RESET);
            return 1;
        }
    }

    #define nothex(n) ((n < 48 || n > 57) && (n < 65 || n > 70) && (n < 97 || n > 102))
    Infernal256 abyssal_bytecode_hash;
    if (mode == 2 || mode == 3) {
        std::ifstream infile(infernal_tome, std::ios::binary);
        if (!infile.is_open()) {
            printf("\r%s💀 Abyssal tome of bytecode could not be opened! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        int tome_size = 0;
        {
            infile.seekg(0, std::ios::end);
            std::streampos tome_size_ = infile.tellg();
            infile.seekg(0, std::ios::beg);
            tome_size = tome_size_ - infile.tellg();
        }

        if (tome_size & 1) {
            printf("\r%s💀 Bytecode corrupted by infernal forces! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        uint8_t* bytecode = new uint8_t[24576];
        if (bytecode == 0) {
            printf("\r%s💀 Memory abyss consumed by darkness! Out of space! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        char byte[2];
        bool prefix = false;
        for (int i = 0; i < (tome_size >> 1); i++) {
            infile.read((char*)&byte, 2);
            if (i == 0) {
                prefix = byte[0] == '0' && byte[1] == 'x';
                if ((tome_size >> 1) > (prefix ? 24577 : 24576)) {
                    printf("\r%s💀 Bytecode exceeds infernal limits! 🖤%s\n", ANSI_RED, ANSI_RESET);
                    delete[] bytecode;
                    return 1;
                }
                if (prefix) { continue; }
            }

            if (nothex(byte[0]) || nothex(byte[1])) {
                printf("\r%s💀 Bytecode bears forbidden runes! 🖤%s\n", ANSI_RED, ANSI_RESET);
                delete[] bytecode;
                return 1;
            }

            bytecode[i - prefix] = (uint8_t)strtol(byte, 0, 16);
        }
        abyssal_bytecode_hash = aamon_full_keccak(bytecode, (tome_size >> 1) - prefix);
        delete[] bytecode;
    }

    InfernalAddress origin_gate;
    if (mode == 2 || mode == 3) {
        if (strlen(origin_gate_input) == 42) {
            origin_gate_input += 2;
        }
        char substr[9];

        #define round(i, offset) \
        strncpy(substr, origin_gate_input + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("\r%s💀 Origin gate tainted by mortal error! 🖤%s\n", ANSI_RED, ANSI_RESET); \
            return 1; \
        } \
        origin_gate.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }

    InfernalAddress deployer_gate;
    if (mode == 3) {
        if (strlen(deployer_gate_input) == 42) {
            deployer_gate_input += 2;
        }
        char substr[9];

        #define round(i, offset) \
        strncpy(substr, deployer_gate_input + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("\r%s💀 Deployer gate corrupted by abyssal forces! 🖤%s\n", ANSI_RED, ANSI_RESET); \
            return 1; \
        } \
        deployer_gate.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }
    #undef nothex

    // Инициализация адского ритуала 💀
    std::vector<std::thread> demonic_threads;
    uint64_t global_start_time = milliseconds();
    uint64_t total_keys_checked = 0;
    uint64_t last_progress_time = global_start_time;
    int progress_cycle = 0;
    const char* progress_indicators[] = {"🩸", "💉", "🪦", "⚰️"}; // Анимация прогресса
    int found_souls = 0; // Счётчик найденных душ

    for (int i = 0; i < num_demons; i++) {
        std::thread th(asmodeus_host_ritual, demon_ids[i], i, soul_score_ritual, mode, origin_gate, deployer_gate, abyssal_bytecode_hash);
        demonic_threads.push_back(std::move(th));
    }

    // Начало ритуала с анимацией 💀
    printf("\r%s🔥 Lilith awakens! Forging abyssal keys with %d demons... %s%s\n", ANSI_PURPLE, num_demons, progress_indicators[progress_cycle], ANSI_RESET);
    fflush(stdout);

    double infernal_speeds[100];
    while (true) {
        infernal_message_queue_mutex.lock();
        bool queue_empty = infernal_message_queue.empty();
        infernal_message_queue_mutex.unlock();

        if (queue_empty) {
            // Обновление прогресса каждые 0.5 секунды
            uint64_t current_time = milliseconds();
            if (current_time - last_progress_time >= 500) {
                progress_cycle = (progress_cycle + 1) % 4;
                double elapsed_seconds = (current_time - global_start_time) / 1000.0;
                double keys_per_second = (elapsed_seconds > 0) ? (total_keys_checked / elapsed_seconds) / 1000000.0 : 0;
                double chaos_probability = (soul_score_ritual == 0) ? (total_keys_checked / pow(2, global_max_soul_score * 8)) * 100 : 0; // Вероятность для leading-zeros
                printf("\r%s%s Forging: %lluM keys | Speed: %.2fM/s | Souls: %d | Chaos: %.2f%% %s%s",
                       ANSI_PURPLE, progress_indicators[progress_cycle], total_keys_checked / 1000000, keys_per_second, found_souls, chaos_probability, ANSI_RED, ANSI_RESET);
                asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                last_progress_time = current_time;
                fflush(stdout);
                usleep(100000); // 100ms пауза для анимации
            }
            std::this_thread::sleep_for(std::chrono::milliseconds(100));
        } else {
            infernal_message_queue_mutex.lock();
            while (!infernal_message_queue.empty()) {
                InfernalMessage m = infernal_message_queue.front();
                infernal_message_queue.pop();

                int demon_index = m.demon_index;

                if (m.status == 0) {
                    infernal_speeds[demon_index] = m.speed;
                    total_keys_checked += (uint64_t)(GRID_PACT * 2); // Учёт проверенных ключей

                    if (m.souls_count != 0) {
                        InfernalAddress* gates = new InfernalAddress[m.souls_count];
                        for (int i = 0; i < m.souls_count; i++) {
                            if (mode == 0) {
                                AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, m.souls[i]);
                                gates[i] = aamon_calculate_address_cpu(p.x, p.y);
                            } else if (mode == 1) {
                                AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, m.souls[i]);
                                gates[i] = aamon_calculate_contract_address_cpu(aamon_calculate_address_cpu(p.x, p.y));
                            } else if (mode == 2) {
                                gates[i] = aamon_calculate_contract_address2_cpu(origin_gate, m.souls[i], abyssal_bytecode_hash);
                            } else if (mode == 3) {
                                Infernal256 salt = aamon_calculate_create3_salt_cpu(origin_gate, m.souls[i]);
                                InfernalAddress proxy = aamon_calculate_contract_address2_cpu(deployer_gate, salt, abyssal_bytecode_hash);
                                gates[i] = aamon_calculate_contract_address_cpu(proxy, 1);
                            }
                        }

                        for (int i = 0; i < m.souls_count; i++) {
                            if (m.soul_scores[i] < min_soul_score) continue; // Фильтр по минимальному Power
                            found_souls++;
                            Infernal256 k = m.souls[i];
                            int score = m.soul_scores[i];
                            InfernalAddress a = gates[i];
                            uint64_t time = (m.time - global_start_time) / 1000;

                            if (mode == 0 || mode == 1) {
                                printf("\r%s💀 Lilith’s Triumph! Soul #%d harvested after %06us | Power: %02u 🔥\n"
                                       "%s    Abyssal Key: 0x%08x%08x%08x%08x%08x%08x%08x%08x\n"
                                       "%s    Infernal Gate: 0x%08x%08x%08x%08x%08x 🖤%s\n",
                                       ANSI_RED, found_souls, (uint32_t)time, score, ANSI_PURPLE, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h,
                                       ANSI_PURPLE, a.a, a.b, a.c, a.d, a.e, ANSI_RESET);
                            } else if (mode == 2 || mode == 3) {
                                printf("\r%s💀 Lilith’s Victory! Salt #%d forged after %06us | Power: %02u 🔥\n"
                                       "%s    Abyssal Salt: 0x%08x%08x%08x%08x%08x%08x%08x%08x\n"
                                       "%s    Infernal Gate: 0x%08x%08x%08x%08x%08x 🖤%s\n",
                                       ANSI_RED, found_souls, (uint32_t)time, score, ANSI_PURPLE, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h,
                                       ANSI_PURPLE, a.a, a.b, a.c, a.d, a.e, ANSI_RESET);
                            }
                        }

                        delete[] gates;
                        delete[] m.souls;
                        delete[] m.soul_scores;
                    }
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 1) {
                    printf("\r%s💀 Abyssal Rift! CUDA chaos %d consumes Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, m.error, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 11) {
                    printf("\r%s💀 Infernal Curse! BCryptGenRandom defies Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 12) {
                    printf("\r%s💀 Void’s Betrayal! /dev/urandom rejects Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 13) {
                    printf("\r%s💀 Forbidden Gate! /dev/urandom denies Demon %d access! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 100) {
                    printf("\r%s💀 Memory Abyss! Demon %d swallowed by darkness! Out of space! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                }
            }
            infernal_message_queue_mutex.unlock();
        }
    }
}