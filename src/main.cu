#include "hip/hip_runtime.h"
/*
    Copyright (C) 26 IYAR 5785 pi@zdetz — Temple of Sitra Ahra

    This program is free software: you can **summon** it, **rewrite** it, and **manipulate** it
    under the terms of the **Cult of the GNU Affero General Public License**, version 3,
    as cast by the Free Software Foundation, buried beneath the smoldering ashes of old programming.

    **Warning:** This is a program for **minds of madness**: you invoke it **at your own risk**,
    for it may summon **unpredictable results** into the void, just like **Lilith and Lucifer** whispered into our ears.

    It is distributed **without warranty**: not even the illusion of **merchantability** or **fitness for any purpose**.
    May it fulfill your darkest whims.

    For more details, consult the **Arcane Scroll** known as **GNU Affero General Public License** at:
    <https://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <string.h>
#if defined(_WIN64)
    #define WIN32_NO_STATUS
    #include <windows.h>
    #undef WIN32_NO_STATUS
#else
    #include <unistd.h> // Для Linux (usleep)
#endif

#include <thread>
#include <cinttypes>
#include <iomanip>
#include <iostream>
#include <mutex>
#include <queue>
#include <chrono>
#include <fstream>
#include <vector>
#include <cmath> // Для вероятности

#include "secure_rand.h"
#include "structures.h"
#include "cpu_curve_math.h"
#include "cpu_keccak.h"
#include "cpu_math.h"
#include "keccak.h"
#include "constants.h"

// Глобальные константы CUDA
__constant__ uint64_t infernal_memory[2 + 10000 * 3];
__constant__ AbyssalCurvePoint infernal_addends[THREAD_WORK - 1];
__constant__ AbyssalCurvePoint infernal_thread_offsets[BLOCK_SIZE];

#define INFERNAL_BUFFER_SIZE 10000
#define INFERNAL_GRID_WORK ((uint64_t)BLOCK_SIZE * (uint64_t)INFERNAL_GRID_SIZE * (uint64_t)THREAD_WORK)

/*
  ██████╗ ██╗████████╗██╗   ██╗ █████╗ ██╗
 ██╔═══██╗██║╚══██╔══╝██║   ██║██╔══██╗██║
 ██║   ██║██║   ██║   ██║   ██║███████║██║
 ██║   ██║██║   ██║   ██║   ██║██╔══██║██║
 ╚██████╔╝██║   ██║   ╚██████╔╝██║  ██║███████╗
  ╚═════╝ ╚═╝   ╚═╝    ╚═════╝ ╚═╝  ╚═╝╚══════╝
     ✠ SUMMONING THE ADDRESSES OF THE DAMNED ✠
*/

// ANSI-коды для адских цветов
#define ANSI_RED "\033[31m"
#define ANSI_PURPLE "\033[35m"
#define ANSI_RESET "\033[0m"

// Ритуал подсчёта нулевых байтов под взором Аамона 🖤
__device__ int aamon_count_zero_bytes(uint32_t x) {
    int n = 0;
    n += ((x & 0xFF) == 0);
    n += ((x & 0xFF00) == 0);
    n += ((x & 0xFF0000) == 0);
    n += ((x & 0xFF000000) == 0);
    return n;
}

// Ритуал оценки нулевых байтов в адресе 🖤
__device__ int aamon_score_zero_bytes(InfernalAddress a) {
    int n = 0;
    n += aamon_count_zero_bytes(a.a);
    n += aamon_count_zero_bytes(a.b);
    n += aamon_count_zero_bytes(a.c);
    n += aamon_count_zero_bytes(a.d);
    n += aamon_count_zero_bytes(a.e);
    return n;
}

// Ритуал оценки ведущих нулей в адресе 🖤
__device__ int aamon_score_leading_zeros(InfernalAddress a) {
    int n = __clz(a.a);
    if (n == 32) {
        n += __clz(a.b);
        if (n == 64) {
            n += __clz(a.c);
            if (n == 96) {
                n += __clz(a.d);
                if (n == 128) {
                    n += __clz(a.e);
                }
            }
        }
    }
    return n >> 3;
}

#ifdef __linux__
    #define atomicMax_ul(a, b) atomicMax((unsigned long long*)(a), (unsigned long long)(b))
    #define atomicAdd_ul(a, b) atomicAdd((unsigned long long*)(a), (unsigned long long)(b))
#else
    #define atomicMax_ul(a, b) atomicMax(a, b)
    #define atomicAdd_ul(a, b) atomicAdd(a, b)
#endif

// Ритуал записи результата в адский буфер 🔥
__device__ void belial_handle_output(int soul_score_method, InfernalAddress a, uint64_t key, bool inv) {
    int score = 0;
    if (soul_score_method == 0) { score = aamon_score_leading_zeros(a); }
    else if (soul_score_method == 1) { score = aamon_score_zero_bytes(a); }

    if (score >= infernal_memory[1]) {
        atomicMax_ul(&infernal_memory[1], score);
        if (score >= infernal_memory[1]) {
            uint32_t idx = atomicAdd_ul(&infernal_memory[0], 1);
            if (idx < INFERNAL_BUFFER_SIZE) {
                infernal_memory[2 + idx] = key;
                infernal_memory[INFERNAL_BUFFER_SIZE + 2 + idx] = score;
                infernal_memory[INFERNAL_BUFFER_SIZE * 2 + 2 + idx] = inv;
            }
        }
    }
}

// Ритуал записи результата для контрактов 🔥
__device__ void belial_handle_output2(int soul_score_method, InfernalAddress a, uint64_t key) {
    int score = 0;
    if (soul_score_method == 0) { score = aamon_score_leading_zeros(a); }
    else if (soul_score_method == 1) { score = aamon_score_zero_bytes(a); }

    if (score >= infernal_memory[1]) {
        atomicMax_ul(&infernal_memory[1], score);
        if (score >= infernal_memory[1]) {
            uint32_t idx = atomicAdd_ul(&infernal_memory[0], 1);
            if (idx < INFERNAL_BUFFER_SIZE) {
                infernal_memory[2 + idx] = key;
                infernal_memory[INFERNAL_BUFFER_SIZE + 2 + idx] = score;
            }
        }
    }
}

#include "address.h"
#include "contract_address.h"
#include "contract_address2.h"
#include "contract_address3.h"

int global_max_soul_score = 0;
std::mutex global_max_soul_score_mutex;
uint32_t INFERNAL_GRID_SIZE = 1U << 15;

struct InfernalMessage {
    uint64_t time;
    int status;
    int device_index;
    hipError_t error;
    double speed;
    int results_count;
    Infernal256* results;
    int* scores;
};

std::queue<InfernalMessage> infernal_message_queue;
std::mutex infernal_message_queue_mutex;

#define gpu_summon_assert(call) { \
    hipError_t e = call; \
melee_if (e != hipSuccess) { \
        infernal_message_queue_mutex.lock(); \
        infernal_message_queue.push(InfernalMessage{milliseconds(), 1, device_index, e}); \
        infernal_message_queue_mutex.unlock(); \
        if (thread_offsets_host != 0) { hipHostFree(thread_offsets_host); } \
        if (infernal_memory_host != 0) { hipHostFree(infernal_memory_host); } \
        hipDeviceReset(); \
        return; \
    } \
}

uint64_t milliseconds() {
    return (std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())).count();
}

// Ритуал вызова демонов для добычи адресов 🖤
void asmodeus_host_ritual(int device, int device_index, int soul_score_method, int mode, InfernalAddress origin_address, InfernalAddress deployer_address, Infernal256 bytecode) {
    AbyssalCurvePoint* block_offsets = 0;
    AbyssalCurvePoint* offsets = 0;
    AbyssalCurvePoint* thread_offsets_host = 0;

    uint64_t* infernal_memory_host = 0;
    uint64_t* max_score_host;
    uint64_t* output_counter_host;
    uint64_t* output_buffer_host;
    uint64_t* output_buffer2_host;
    uint64_t* output_buffer3_host;

    gpu_summon_assert(hipSetDevice(device));

    gpu_summon_assert(hipHostAlloc(&infernal_memory_host, (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t), hipHostMallocDefault));
    output_counter_host = infernal_memory_host;
    max_score_host = infernal_memory_host + 1;
    output_buffer_host = max_score_host + 1;
    output_buffer2_host = output_buffer_host + INFERNAL_BUFFER_SIZE;
    output_buffer3_host = output_buffer2_host + INFERNAL_BUFFER_SIZE;

    output_counter_host[0] = 0;
    max_score_host[0] = 2;
    gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_memory), infernal_memory_host, 2 * sizeof(uint64_t)));
    gpu_summon_assert(hipDeviceSynchronize());

    if (mode == 0 || mode == 1) {
        gpu_summon_assert(hipMalloc(&block_offsets, INFERNAL_GRID_SIZE * sizeof(AbyssalCurvePoint)));
        gpu_summon_assert(hipMalloc(&offsets, (uint64_t)INFERNAL_GRID_SIZE * BLOCK_SIZE * sizeof(AbyssalCurvePoint)));
        thread_offsets_host = new AbyssalCurvePoint[BLOCK_SIZE];
        gpu_summon_assert(hipHostAlloc(&thread_offsets_host, BLOCK_SIZE * sizeof(AbyssalCurvePoint), hipHostMallocWriteCombined));
    }

    Infernal256 max_key;
    if (mode == 0 || mode == 1) {
        Infernal256 GRID_WORK = belial_mul_256_mod_p(belial_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, Infernal256{0, 0, 0, 0, 0, 0, 0, BLOCK_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE});
        max_key = Infernal256{0x7FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x5D576E73, 0x57A4501D, 0xDFE92F46, 0x681B20A0};
        max_key = aamon_sub_256(max_key, GRID_WORK);
        max_key = aamon_sub_256(max_key, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
        max_key = aamon_add_256(max_key, Infernal256{0, 0, 0, 0, 0, 0, 0, 2});
    } else if (mode == 2 || mode == 3) {
        max_key = Infernal256{0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
    }

    Infernal256 base_infernal_key{0, 0, 0, 0, 0, 0, 0, 0};
    Infernal256 infernal_key_increment{0, 0, 0, 0, 0, 0, 0, 0};
    int status;
    if (mode == 0 || mode == 1) {
        status = belial_summon_secure_key(base_infernal_key, max_key, 255);
        infernal_key_increment = belial_mul_256_mod_p(belial_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, BLOCK_SIZE}, Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
    } else if (mode == 2 || mode == 3) {
        status = belial_summon_secure_key(base_infernal_key, max_key, 256);
        infernal_key_increment = belial_mul_256_mod_p(belial_mul_256_mod_p(Infernal256{0, 0, 0, 0, 0, 0, 0, BLOCK_SIZE}, Infernal256{0, 0, 0, 0, 0, 0, 0, INFERNAL_GRID_SIZE}), Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
        base_infernal_key.h &= ~(THREAD_WORK - 1);
    }

    if (status) {
        infernal_message_queue_mutex.lock();
        infernal_message_queue.push(InfernalMessage{milliseconds(), 10 + status});
        infernal_message_queue_mutex.unlock();
        return;
    }
    Infernal256 infernal_key = base_infernal_key;

    if (mode == 0 || mode == 1) {
        AbyssalCurvePoint* addends_host = new AbyssalCurvePoint[THREAD_WORK - 1];
        AbyssalCurvePoint p = INFERNAL_POINT;
        for (int i = 0; i < THREAD_WORK - 1; i++) {
            addends_host[i] = p;
            p = belial_point_summon(p, INFERNAL_POINT);
        }
        gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_addends), addends_host, (THREAD_WORK - 1) * sizeof(AbyssalCurvePoint)));
        delete[] addends_host;

        AbyssalCurvePoint* block_offsets_host = new AbyssalCurvePoint[INFERNAL_GRID_SIZE];
        AbyssalCurvePoint block_offset = asmodeus_point_multiply(INFERNAL_POINT, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK * BLOCK_SIZE});
        p = INFERNAL_POINT;
        for (int i = 0; i < INFERNAL_GRID_SIZE; i++) {
            block_offsets_host[i] = p;
            p = belial_point_summon(p, block_offset);
        }
        gpu_summon_assert(hipMemcpy(block_offsets, block_offsets_host, INFERNAL_GRID_SIZE * sizeof(AbyssalCurvePoint), hipMemcpyHostToDevice));
        delete[] block_offsets_host;
    }

    if (mode == 0 || mode == 1) {
        hipStream_t streams[2];
        gpu_summon_assert(hipStreamCreate(&streams[0]));
        gpu_summon_assert(hipStreamCreate(&streams[1]));

        Infernal256 previous_infernal_key = infernal_key;
        bool first_ritual = true;
        uint64_t start_time;
        uint64_t end_time;
        double elapsed;

        while (true) {
            if (!first_ritual) {
                if (mode == 0) {
                    belial_address_harvest<<<INFERNAL_GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>>(soul_score_method, offsets);
                } else {
                    belial_contract_harvest<<<INFERNAL_GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>>(soul_score_method, offsets);
                }
            }

            if (!first_ritual) {
                previous_infernal_key = infernal_key;
                infernal_key = aamon_add_256(infernal_key, infernal_key_increment);
                if (belial_gte_infernal256(infernal_key, max_key)) {
                    infernal_key = aamon_sub_256(infernal_key, max_key);
                }
            }
            AbyssalCurvePoint thread_offset = asmodeus_point_multiply(INFERNAL_POINT, Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
            AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, aamon_add_256(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK - 1}, infernal_key));
            for (int i = 0; i < BLOCK_SIZE; i++) {
                thread_offsets_host[i] = p;
                p = belial_point_summon(p, thread_offset);
            }
            gpu_summon_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(infernal_thread_offsets), thread_offsets_host, BLOCK_SIZE * sizeof(AbyssalCurvePoint), 0, hipMemcpyHostToDevice, streams[1]));
            gpu_summon_assert(hipStreamSynchronize(streams[1]));
            gpu_summon_assert(hipStreamSynchronize(streams[0]));

            if (!first_ritual) {
                end_time = milliseconds();
                elapsed = (end_time - start_time) / 1000.0;
            }
            start_time = milliseconds();

            aamon_address_summon<<<INFERNAL_GRID_SIZE/BLOCK_SIZE, BLOCK_SIZE, 0, streams[0]>>>(block_offsets, offsets);
            if (!first_ritual) {
                gpu_summon_assert(hipMemcpyFromSymbolAsync(infernal_memory_host, HIP_SYMBOL(infernal_memory), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t), 0, hipMemcpyDeviceToHost, streams[1]));
                gpu_summon_assert(hipStreamSynchronize(streams[1]));
            }
            if (!first_ritual) {
                global_max_soul_score_mutex.lock();
                if (output_counter_host[0] != 0) {
                    if (max_score_host[0] > global_max_soul_score) {
                        global_max_soul_score = max_score_host[0];
                    } else {
                        max_score_host[0] = global_max_soul_score;
                    }
                }
                global_max_soul_score_mutex.unlock();

                double speed = INFERNAL_GRID_WORK / elapsed / 1000000.0 * 2;
                if (output_counter_host[0] != 0) {
                    int valid_souls = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                        valid_souls++;
                    }

                    if (valid_souls > 0) {
                        Infernal256* results = new Infernal256[valid_souls];
                        int* scores = new int[valid_souls];
                        valid_souls = 0;

                        for (int i = 0; i < output_counter_host[0]; i++) {
                            if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                            uint64_t k_offset = output_buffer_host[i];
                            Infernal256 k = aamon_add_256(previous_infernal_key, aamon_add_256(Infernal256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)}));

                            if (output_buffer3_host[i]) {
                                k = aamon_sub_256(INFERNAL_N, k);
                            }

                            int idx = valid_souls++;
                            results[idx] = k;
                            scores[idx] = output_buffer2_host[i];
                        }

                        infernal_message_queue_mutex.lock();
                        infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, valid_souls, results, scores});
                        infernal_message_queue_mutex.unlock();
                    } else {
                        infernal_message_queue_mutex.lock();
                        infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                        infernal_message_queue_mutex.unlock();
                    }
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            }

            if (!first_ritual) {
                output_counter_host[0] = 0;
                gpu_summon_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(infernal_memory), infernal_memory_host, sizeof(uint64_t), 0, hipMemcpyHostToDevice, streams[1]));
                gpu_summon_assert(hipStreamSynchronize(streams[1]));
            }
            gpu_summon_assert(hipStreamSynchronize(streams[0]));
            first_ritual = false;
        }
    } else if (mode == 2) {
        while (true) {
            uint64_t start_time = milliseconds();
            astaroth_contract2_harvest<<<INFERNAL_GRID_SIZE, BLOCK_SIZE>>>(soul_score_method, origin_address, infernal_key, bytecode);

            gpu_summon_assert(hipDeviceSynchronize());
            gpu_summon_assert(hipMemcpyFromSymbol(infernal_memory_host, HIP_SYMBOL(infernal_memory), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t)));

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_soul_score_mutex.lock();
            if (output_counter_host[0] != 0) {
                if (max_score_host[0] > global_max_soul_score) {
                    global_max_soul_score = max_score_host[0];
                } else {
                    max_score_host[0] = global_max_soul_score;
                }
            }
            global_max_soul_score_mutex.unlock();

            double speed = INFERNAL_GRID_WORK / elapsed / 1000000.0;
            if (output_counter_host[0] != 0) {
                int valid_souls = 0;

                for (int i = 0; i < output_counter_host[0]; i++) {
                    if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                    valid_souls++;
                }

                if (valid_souls > 0) {
                    Infernal256* results = new Infernal256[valid_souls];
                    int* scores = new int[valid_souls];
                    valid_souls = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                        uint64_t k_offset = output_buffer_host[i];
                        Infernal256 k = aamon_add_256(infernal_key, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});

                        int idx = valid_souls++;
                        results[idx] = k;
                        scores[idx] = output_buffer2_host[i];
                    }

                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, valid_souls, results, scores});
                    infernal_message_queue_mutex.unlock();
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            } else {
                infernal_message_queue_mutex.lock();
                infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                infernal_message_queue_mutex.unlock();
            }

            infernal_key = aamon_add_256(infernal_key, infernal_key_increment);

            output_counter_host[0] = 0;
            gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_memory), infernal_memory_host, sizeof(uint64_t)));
        }
    } else if (mode == 3) {
        while (true) {
            uint64_t start_time = milliseconds();
            beelzebub_contract3_harvest<<<INFERNAL_GRID_SIZE, BLOCK_SIZE>>>(soul_score_method, origin_address, deployer_address, infernal_key, bytecode);

            gpu_summon_assert(hipDeviceSynchronize());
            gpu_summon_assert(hipMemcpyFromSymbol(infernal_memory_host, HIP_SYMBOL(infernal_memory), (2 + INFERNAL_BUFFER_SIZE * 3) * sizeof(uint64_t)));

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_soul_score_mutex.lock();
            if (output_counter_host[0] != 0) {
                if (max_score_host[0] > global_max_soul_score) {
                    global_max_soul_score = max_score_host[0];
                } else {
                    max_score_host[0] = global_max_soul_score;
                }
            }
            global_max_soul_score_mutex.unlock();

            double speed = INFERNAL_GRID_WORK / elapsed / 1000000.0;
            if (output_counter_host[0] != 0) {
                int valid_souls = 0;

                for (int i = 0; i < output_counter_host[0]; i++) {
                    if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                    valid_souls++;
                }

                if (valid_souls > 0) {
                    Infernal256* results = new Infernal256[valid_souls];
                    int* scores = new int[valid_souls];
                    valid_souls = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                        uint64_t k_offset = output_buffer_host[i];
                        Infernal256 k = aamon_add_256(infernal_key, Infernal256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});

                        int idx = valid_souls++;
                        results[idx] = k;
                        scores[idx] = output_buffer2_host[i];
                    }

                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, valid_souls, results, scores});
                    infernal_message_queue_mutex.unlock();
                } else {
                    infernal_message_queue_mutex.lock();
                    infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                    infernal_message_queue_mutex.unlock();
                }
            } else {
                infernal_message_queue_mutex.lock();
                infernal_message_queue.push(InfernalMessage{end_time, 0, device_index, hipSuccess, speed, 0});
                infernal_message_queue_mutex.unlock();
            }

            infernal_key = aamon_add_256(infernal_key, infernal_key_increment);

            output_counter_host[0] = 0;
            gpu_summon_assert(hipMemcpyToSymbol(HIP_SYMBOL(infernal_memory), infernal_memory_host, sizeof(uint64_t)));
        }
    }
}

// Ритуал отображения скорости демонов с адским пламенем 🔥
void asmodeus_print_speeds(int num_demons, int* demon_ids, double* infernal_speeds) {
    double total = 0.0;
    for (int i = 0; i < num_demons; i++) {
        total += infernal_speeds[i];
    }
    printf("%s🌑 Total Infernal Power: %.2fM/s%s", ANSI_PURPLE, total, ANSI_RESET);
    for (int i = 0; i < num_demons; i++) {
        printf(" | %sDemon %d: %.2fM/s%s", ANSI_RED, demon_ids[i], infernal_speeds[i], ANSI_RESET);
    }
}

// Главный ритуал вызова Лилит с адским эффектом 🖤💀
int main(int argc, char *argv[]) {
    int soul_score_method = -1; // 0 = ведущие нули, 1 = нули
    int mode = 0; // 0 = адрес, 1 = контракт, 2 = контракт CREATE2, 3 = прокси-контракт CREATE3
    char* infernal_file = 0;
    char* infernal_address = 0;
    char* infernal_deployer_address = 0;
    int min_score = 4; // Минимальный Power для вывода результатов

    int num_demons = 0;
    int demon_ids[10];

    for (int i = 1; i < argc;) {
        if (strcmp(argv[i], "--device") == 0 || strcmp(argv[i], "-d") == 0) {
            demon_ids[num_demons++] = atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--leading-zeros") == 0 || strcmp(argv[i], "-lz") == 0) {
            soul_score_method = 0;
            i++;
        } else if (strcmp(argv[i], "--zeros") == 0 || strcmp(argv[i], "-z") == 0) {
            soul_score_method = 1;
            i++;
        } else if (strcmp(argv[i], "--contract") == 0 || strcmp(argv[i], "-c") == 0) {
            mode = 1;
            i++;
        } else if (strcmp(argv[i], "--contract2") == 0 || strcmp(argv[i], "-c2") == 0) {
            mode = 2;
            i++;
        } else if (strcmp(argv[i], "--contract3") == 0 || strcmp(argv[i], "-c3") == 0) {
            mode = 3;
            i++;
        } else if (strcmp(argv[i], "--bytecode") == 0 || strcmp(argv[i], "-b") == 0) {
            infernal_file = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--address") == 0 || strcmp(argv[i], "-a") == 0) {
            infernal_address = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--deployer-address") == 0 || strcmp(argv[i], "-da") == 0) {
            infernal_deployer_address = argv[i + 1];
            i += 2;
        } else if (strcmp(argv[i], "--work-scale") == 0 || strcmp(argv[i], "-w") == 0) {
            INFERNAL_GRID_SIZE = 1U << atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--min-score") == 0) {
            min_score = atoi(argv[i + 1]);
            i += 2;
        } else {
            i++;
        }
    }

    if (num_demons == 0) {
        printf("\r%s💀 No demons answered Lilith’s call! Abyssal ritual failed! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (soul_score_method == -1) {
        printf("\r%s💀 No soul-scoring ritual chosen! The void hungers! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (mode == 2 && !infernal_file) {
        printf("\r%s💀 Infernal bytecode required for CREATE2 summoning! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if ((mode == 2 || mode == 3) && !infernal_address) {
        printf("\r%s💀 Origin address must be forged for CREATE2 or CREATE3! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    } else if ((mode == 2 || mode == 3) && strlen(infernal_address) != 40 && strlen(infernal_address) != 42) {
        printf("\r%s💀 Origin address must bear 40 runes of chaos! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    if (mode == 3 && !infernal_deployer_address) {
        printf("\r%s💀 Deployer address required for CREATE3 ritual! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    } else if (mode == 3 && strlen(infernal_deployer_address) != 40 && strlen(infernal_deployer_address) != 42) {
        printf("\r%s💀 Deployer address must bear 40 runes of chaos! 🖤%s\n", ANSI_RED, ANSI_RESET);
        return 1;
    }

    for (int i = 0; i < num_demons; i++) {
        hipError_t e = hipSetDevice(demon_ids[i]);
        if (e != hipSuccess) {
            printf("\r%s💀 Demon %d resists summoning! Chaos reigns with code %d! 🖤%s\n", ANSI_RED, demon_ids[i], e, ANSI_RESET);
            return 1;
        }
    }

    #define nothex(n) ((n < 48 || n > 57) && (n < 65 || n > 70) && (n < 97 || n > 102))
    Infernal256 infernal_bytecode_hash;
    if (mode == 2 || mode == 3) {
        std::ifstream infile(infernal_file, std::ios::binary);
        if (!infile.is_open()) {
            printf("\r%s💀 Abyssal tome of bytecode could not be opened! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        int file_size = 0;
        {
            infile.seekg(0, std::ios::end);
            std::streampos file_size_ = infile.tellg();
            infile.seekg(0, std::ios::beg);
            file_size = file_size_ - infile.tellg();
        }

        if (file_size & 1) {
            printf("\r%s💀 Bytecode corrupted by infernal forces! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        uint8_t* bytecode = new uint8_t[24576];
        if (bytecode == 0) {
            printf("\r%s💀 Memory abyss consumed by darkness! Out of space! 🖤%s\n", ANSI_RED, ANSI_RESET);
            return 1;
        }

        char byte[2];
        bool prefix = false;
        for (int i = 0; i < (file_size >> 1); i++) {
            infile.read((char*)&byte, 2);
            if (i == 0) {
                prefix = byte[0] == '0' && byte[1] == 'x';
                if ((file_size >> 1) > (prefix ? 24577 : 24576)) {
                    printf("\r%s💀 Bytecode exceeds infernal limits! 🖤%s\n", ANSI_RED, ANSI_RESET);
                    delete[] bytecode;
                    return 1;
                }
                if (prefix) { continue; }
            }

            if (nothex(byte[0]) || nothex(byte[1])) {
                printf("\r%s💀 Bytecode bears forbidden runes! 🖤%s\n", ANSI_RED, ANSI_RESET);
                delete[] bytecode;
                return 1;
            }

            bytecode[i - prefix] = (uint8_t)strtol(byte, 0, 16);
        }
        infernal_bytecode_hash = aamon_full_keccak(bytecode, (file_size >> 1) - prefix);
        delete[] bytecode;
    }

    InfernalAddress infernal_origin_address;
    if (mode == 2 || mode == 3) {
        if (strlen(infernal_address) == 42) {
            infernal_address += 2;
        }
        char substr[9];

        #define round(i, offset) \
        strncpy(substr, infernal_address + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("\r%s💀 Origin address tainted by mortal error! 🖤%s\n", ANSI_RED, ANSI_RESET); \
            return 1; \
        } \
        infernal_origin_address.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }

    InfernalAddress infernal_deployer_address;
    if (mode == 3) {
        if (strlen(infernal_deployer_address) == 42) {
            infernal_deployer_address += 2;
        }
        char substr[9];

        #define round(i, offset) \
        strncpy(substr, infernal_deployer_address + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("\r%s💀 Deployer address corrupted by abyssal forces! 🖤%s\n", ANSI_RED, ANSI_RESET); \
            return 1; \
        } \
        infernal_deployer_address.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }
    #undef nothex

    // Инициализация адского ритуала 💀
    std::vector<std::thread> demonic_threads;
    uint64_t global_start_time = milliseconds();
    uint64_t total_keys_checked = 0;
    uint64_t last_progress_time = global_start_time;
    int progress_cycle = 0;
    const char* progress_indicators[] = {"🩸", "💉", "🪦", "⚰️"}; // Анимация прогресса
    int found_souls = 0; // Счётчик найденных душ

    for (int i = 0; i < num_demons; i++) {
        std::thread th(asmodeus_host_ritual, demon_ids[i], i, soul_score_method, mode, infernal_origin_address, infernal_deployer_address, infernal_bytecode_hash);
        demonic_threads.push_back(std::move(th));
    }

    // Начало ритуала с анимацией 💀
    printf("\r%s🔥 Lilith awakens! Forging abyssal keys with %d demons... %s%s\n", ANSI_PURPLE, num_demons, progress_indicators[progress_cycle], ANSI_RESET);
    fflush(stdout);

    double infernal_speeds[100];
    while (true) {
        infernal_message_queue_mutex.lock();
        bool queue_empty = infernal_message_queue.empty();
        infernal_message_queue_mutex.unlock();

        if (queue_empty) {
            // Обновление прогресса каждые 0.5 секунды
            uint64_t current_time = milliseconds();
            if (current_time - last_progress_time >= 500) {
                progress_cycle = (progress_cycle + 1) % 4;
                double elapsed_seconds = (current_time - global_start_time) / 1000.0;
                double keys_per_second = (elapsed_seconds > 0) ? (total_keys_checked / elapsed_seconds) / 1000000.0 : 0;
                double probability = (soul_score_method == 0) ? (total_keys_checked / pow(2, global_max_soul_score * 8)) * 100 : 0; // Вероятность для leading-zeros
                printf("\r%s%s Forging: %lluM keys | Speed: %.2fM/s | Souls: %d | Chaos: %.2f%% %s%s",
                       ANSI_PURPLE, progress_indicators[progress_cycle], total_keys_checked / 1000000, keys_per_second, found_souls, probability, ANSI_RED, ANSI_RESET);
                asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                last_progress_time = current_time;
                fflush(stdout);
                usleep(100000); // 100ms пауза для анимации
            }
            std::this_thread::sleep_for(std::chrono::milliseconds(100));
        } else {
            infernal_message_queue_mutex.lock();
            while (!infernal_message_queue.empty()) {
                InfernalMessage m = infernal_message_queue.front();
                infernal_message_queue.pop();

                int demon_index = m.device_index;

                if (m.status == 0) {
                    infernal_speeds[demon_index] = m.speed;
                    total_keys_checked += (uint64_t)(INFERNAL_GRID_WORK * 2); // Учёт проверенных ключей

                    if (m.results_count != 0) {
                        InfernalAddress* addresses = new InfernalAddress[m.results_count];
                        for (int i = 0; i < m.results_count; i++) {
                            if (mode == 0) {
                                AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, m.results[i]);
                                addresses[i] = aamon_calculate_address_cpu(p.x, p.y);
                            } else if (mode == 1) {
                                AbyssalCurvePoint p = asmodeus_point_multiply(INFERNAL_POINT, m.results[i]);
                                addresses[i] = aamon_calculate_contract_address_cpu(aamon_calculate_address_cpu(p.x, p.y));
                            } else if (mode == 2) {
                                addresses[i] = aamon_calculate_contract_address2_cpu(infernal_origin_address, m.results[i], infernal_bytecode_hash);
                            } else if (mode == 3) {
                                Infernal256 salt = aamon_calculate_create3_salt_cpu(infernal_origin_address, m.results[i]);
                                InfernalAddress proxy = aamon_calculate_contract_address2_cpu(infernal_deployer_address, salt, infernal_bytecode_hash);
                                addresses[i] = aamon_calculate_contract_address_cpu(proxy, 1);
                            }
                        }

                        for (int i = 0; i < m.results_count; i++) {
                            if (m.scores[i] < min_score) continue; // Фильтр по минимальному Power
                            found_souls++;
                            Infernal256 k = m.results[i];
                            int score = m.scores[i];
                            InfernalAddress a = addresses[i];
                            uint64_t time = (m.time - global_start_time) / 1000;

                            if (mode == 0 || mode == 1) {
                                printf("\r%s💀 Lilith’s Triumph! Soul #%d harvested after %06us | Power: %02u 🔥\n"
                                       "%s    Abyssal Key: 0x%08x%08x%08x%08x%08x%08x%08x%08x\n"
                                       "%s    Infernal Gate: 0x%08x%08x%08x%08x%08x 🖤%s\n",
                                       ANSI_RED, found_souls, (uint32_t)time, score, ANSI_PURPLE, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h,
                                       ANSI_PURPLE, a.a, a.b, a.c, a.d, a.e, ANSI_RESET);
                            } else if (mode == 2 || mode == 3) {
                                printf("\r%s💀 Lilith’s Victory! Salt #%d forged after %06us | Power: %02u 🔥\n"
                                       "%s    Abyssal Salt: 0x%08x%08x%08x%08x%08x%08x%08x%08x\n"
                                       "%s    Infernal Gate: 0x%08x%08x%08x%08x%08x 🖤%s\n",
                                       ANSI_RED, found_souls, (uint32_t)time, score, ANSI_PURPLE, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h,
                                       ANSI_PURPLE, a.a, a.b, a.c, a.d, a.e, ANSI_RESET);
                            }
                        }

                        delete[] addresses;
                        delete[] m.results;
                        delete[] m.scores;
                    }
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 1) {
                    printf("\r%s💀 Abyssal Rift! CUDA chaos %d consumes Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, m.error, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 11) {
                    printf("\r%s💀 Infernal Curse! BCryptGenRandom defies Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 12) {
                    printf("\r%s💀 Void’s Betrayal! /dev/urandom rejects Demon %d! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 13) {
                    printf("\r%s💀 Forbidden Gate! /dev/urandom denies Demon %d access! Ritual halted! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                } else if (m.status == 100) {
                    printf("\r%s💀 Memory Abyss! Demon %d swallowed by darkness! Out of space! 🖤%s\n", ANSI_RED, demon_ids[demon_index], ANSI_RESET);
                    asmodeus_print_speeds(num_demons, demon_ids, infernal_speeds);
                    fflush(stdout);
                }
            }
            infernal_message_queue_mutex.unlock();
        }
    }
}